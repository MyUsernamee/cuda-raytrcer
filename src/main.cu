#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <GL/gl.h>
#include <math_functions.h>
#include "math.h"
#include "intersections.h"
#include <glm/gtc/matrix_transform.hpp>
#include <sstream>
#include <hiprand.h>

__host__ __device__ float rand(glm::vec3 co)
{
    float a_;
    return modf((float)(sin(dot(co, glm::vec3(12415.9898, 7318.233, 16126.2512))) * 43758.5453), &a_);
}

__device__ __host__ void
setColor(unsigned char *image, glm::vec4 color, int x, int y, int width, int height)
{

    image[((y * width) + x) * 4] = max(min(color.x * 255, 255.0), 0.0);
    image[((y * width) + x) * 4 + 1] = max(min(color.y * 255, 255.0), 0.0);
    image[((y * width) + x) * 4 + 2] = max(min(color.z * 255, 255.0), 0.0);
    image[((y * width) + x) * 4 + 3] = max(min(color.w * 255, 255.0), 0.0);
}

__host__ __device__ glm::vec3 sample(float u1, float u2)
{

    float r = sqrt(u1);
    float theta = 2 * M_PI * u2;

    float x = r * cos(theta);
    float y = r * sin(theta);

    return glm::vec3(x, y, sqrt(max(0.0f, 1.0 - u1)));
}

__host__ __device__ glm::vec3 rotate_towards(glm::vec3 reference, glm::vec3 target)
{

    if (reference.y == 1.0)
    {
        return glm::vec3(1.0, 0.0, 0.0) * target.y + glm::vec3(0.0, 1.0, 0.0) * target.x + reference * target.z;
    }

    glm::vec3 right = glm::cross(reference, glm::vec3(0.0, 1.0, 0.0));
    glm::vec3 up = glm::cross(right, reference);

    return right * target.x + up * target.y + reference * target.z;
}

__host__ __device__ glm::vec3 trace(glm::vec3 start, glm::vec3 direction, Object *objects, size_t num_objects, float seed)
{

    glm::vec3 accumulated_light = glm::vec3(0.0);
    glm::vec3 bounced_light = glm::vec3(1.0);

    for (int bounce = 0; bounce < 8; ++bounce)
    {
        auto hit = intersectMany(objects, num_objects, start, direction);

        if (!hit.hit)
            break;

        bounced_light *= hit.object->color;

        auto shadow_hit = intersectMany(objects, num_objects, hit.position + hit.normal * 0.00001f, glm::vec3(0.0, 1.0, 0.0));

        accumulated_light += bounced_light * (shadow_hit.hit ? 0.0f : glm::dot(hit.normal, glm::vec3(0.0, 1.0, 0.0)));

        start = hit.position;
        direction = rotate_towards(hit.normal, sample(rand(hit.position + seed), rand(glm::vec3(hit.position.x + seed, rand(hit.position + seed), hit.position.z))));
    }

    return accumulated_light;
}

__host__ __device__ void render(unsigned char *image, glm::mat4 view_matrix, Object *objects, size_t num_objects, int x, int y, int width, int height, double time)
{

    double aspect_ratio = (float)width / height;

    double x_scaled = ((float)x / (float)width - 0.5) * 2 * aspect_ratio;
    double y_scaled = ((float)y / (float)height - 0.5) * 2;

    glm::vec3 direction = glm::normalize(glm::mat3(glm::inverse(view_matrix)) * glm::vec3(-x_scaled, -y_scaled, -1.0));
    glm::vec3 start = glm::vec3(view_matrix[3]);

    glm::vec3 color = glm::vec3(0.0);

    for (int i = 0; i < 10; i++)
    {

        color += trace(start, direction, objects, num_objects, i + time);
    }

    setColor(image, glm::vec4(color / 10.0f, 1.0), x, y, width, height);
}

__global__ void makeWhite(unsigned char *image, glm::mat4 view_matrix, Object *objects, size_t num_objects, int width, int height, double time)
{

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width || y > height)
        return;

    render(image, view_matrix, objects, num_objects, x, y, width, height, time);
}

int main(int argc, char **argv)
{

    hipSetDevice(0);

    sf::Clock clock;
    int width = 1280;
    int height = 720;

    sf::RenderWindow window(sf::VideoMode(width, height), "Hello!");
    sf::Texture render_texture;
    render_texture.create(width, height);

    window.clear(sf::Color(255, 255, 255, 255));
    window.display();

    sf::Sprite image(render_texture);
    image.setPosition(sf::Vector2f(width / 2, height / 2));
    image.setOrigin(sf::Vector2f(render_texture.getSize()) / 2.f);

    render_texture.update(window);

    size_t image_size = width * height * sizeof(char) * 4;

    // Do something
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x, (height + threadsperBlock.y - 1) / threadsperBlock.y);

    unsigned char *h_image = new unsigned char[image_size];
    unsigned char *d_image;
    hipMalloc(&d_image, image_size);

    glm::mat4 view_matrix = glm::lookAt(glm::vec3(0.0, 0.0, -3.0), glm::vec3(0.0), glm::vec3(0.0, 1.0, 0.0));

    float _;
    glm::vec3 _2;

    static const size_t num_objects = 2;

    Object *objects = new Object[num_objects];

    objects[0] = Object{
        glm::mat4(1.0),
        glm::vec3(1.0, 0.1, 0.1),
        false,
        ObjectType::Sphere};
    objects[1] = Object{
        glm::translate(glm::mat4(1.0), glm::vec3(0.0, -1.0, 0.0)),
        glm::vec3(1.0),
        false,
        ObjectType::Plane};

    Object *d_objects;
    hipMalloc(&d_objects, sizeof(Object) * num_objects);

    hipMemcpy(d_objects, objects, num_objects * sizeof(Object), hipMemcpyHostToDevice);

    float last_x = 0;
    float last_y = 0;

    sf::Clock fps_timer;
    double last_time = 0.0;

    sf::Font font;
    font.loadFromFile("/usr/share/fonts/opentype/roboto/slab/RobotoSlab-Blod.otd");

    window.setMouseCursorGrabbed(true);
    sf::Mouse::setPosition(window.getPosition() + sf::Vector2i(window.getSize()) / 2);

    while (window.isOpen())
    {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();

            if (event.type == sf::Event::MouseMoved)
            {

                view_matrix = glm::rotate(view_matrix, -((float)event.mouseMove.x - (float)last_x) * 0.005f, glm::vec3(0.0, 1.0, 0.0));
                view_matrix = glm::rotate(view_matrix, ((float)event.mouseMove.y - (float)last_y) * 0.005f, glm::vec3(glm::inverse(view_matrix)[0]));

                last_x = event.mouseMove.x;
                last_y = event.mouseMove.y;
            }
        }

        window.clear();

        std::cout << 1.0f / (float)last_time << std::endl;

        makeWhite<<<numBlocks, threadsperBlock>>>(d_image, view_matrix, d_objects, num_objects, width, height, clock.getElapsedTime().asSeconds());

        // for (int x = 0; x < width; ++x)
        // {
        //     for (int y = 0; y < height; ++y)
        //     {
        //         render(h_image, view_matrix, objects, num_objects, x, y, width, height);
        //     }
        // }

        hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);

        // memset(h_image, (unsigned char)255, image_size);

        render_texture.update(h_image);

        window.draw(image);

        window.display();

        last_time = fps_timer.getElapsedTime().asSeconds();
        fps_timer.restart();
    }

    return 0;
}
