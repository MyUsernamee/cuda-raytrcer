#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <GL/gl.h>
#include <math_functions.h>
#include "math.h"
#include "intersections.h"
#include <glm/gtc/matrix_transform.hpp>
#include <sstream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>

__device__ float rand(hiprandState *state)
{
    return hiprand_uniform(state);
}

__global__ void init_curand_state(hiprandState *states, int numBlocks, int seed)
{

    int id = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y + blockDim.y * blockIdx.y) * numBlocks * blockDim.x;

    hiprand_init(seed, id, 0, &states[id]);
};

__device__ glm::vec3 rand_vec(hiprandState_t *state)
{
    return glm::vec3(hiprand_uniform(state), hiprand_uniform(state), hiprand_uniform(state));
}

__device__ __host__ void setColor(unsigned char *image, glm::vec4 color, int x, int y, int width, int height)
{

    image[((y * width) + x) * 4] = max(min(color.x * 255.0, 255.0), 0.0);
    image[((y * width) + x) * 4 + 1] = max(min(color.y * 255.0, 255.0), 0.0);
    image[((y * width) + x) * 4 + 2] = max(min(color.z * 255.0, 255.0), 0.0);
    image[((y * width) + x) * 4 + 3] = max(min(color.w * 255.0, 255.0), 0.0);
}

__host__ __device__ glm::vec3 sample(float u1, float u2)
{

    float r = sqrt(u1);
    float theta = 2 * M_PI * u2;

    float x = r * cos(theta);
    float y = r * sin(theta);

    return glm::vec3(x, y, sqrt(max(0.0f, 1.0 - u1)));
}

__host__ __device__ glm::vec3 reflect(glm::vec3 incoming, glm::vec3 normal)
{
    return -dot(incoming, normal) * normal * 2.0f + incoming;
}

__host__ __device__ glm::vec3 rotate_towards(glm::vec3 reference, glm::vec3 target)
{

    if (reference.y == 1.0)
    {
        return glm::vec3(1.0, 0.0, 0.0) * target.y + glm::vec3(0.0, 1.0, 0.0) * target.x + reference * target.z;
    }

    glm::vec3 right = glm::cross(reference, glm::vec3(0.0, 1.0, 0.0));
    glm::vec3 up = glm::cross(right, reference);

    return right * target.x + up * target.y + reference * target.z;
}

__device__ glm::vec3 generateSample(Object *object, glm::vec3 incoming, glm::vec3 normal, hiprandState *state)
{

    glm::vec3 diffuse = rotate_towards(normal, sample(rand(state), rand(state)));
    glm::vec3 reflection = reflect(incoming, normal);

    return reflection * (1.0f - object->roughness) + diffuse * (object->roughness);
}

__device__ float pdf(glm::vec3 incoming, glm::vec3 normal, glm::vec3 outgoing, float roughness)
{

    glm::vec3 true_outgoing = reflect(incoming, normal);

    float area = roughness;
    float max_angle = (M_PI - M_PI * (1 - roughness));

    float angle = acos(dot(outgoing, true_outgoing));

    return (angle < max_angle) ? (1.0f / area) : 0.0f;
}

__device__ glm::vec3 trace(glm::vec3 start, glm::vec3 direction, Object *objects, size_t *lights, size_t num_lights, size_t num_objects, hiprandState *state)
{

    glm::vec3 accumulated_light = glm::vec3(0.0);
    glm::vec3 bounced_light = glm::vec3(1.0);

    for (int bounce = 0; bounce < 3; ++bounce)
    {
        auto hit = intersectMany(objects, num_objects, start, direction);

        if (!hit.hit)
            break;

        if (hit.object->is_light)
        {
            accumulated_light = bounce == 0 ? bounced_light : accumulated_light;

            break;
        }

        bounced_light *= hit.object->color;

        glm::vec3 light = glm::vec3(0.0);

        for (int light_index = 0; light_index < num_lights; light_index++)
        {

            auto light_position = rand_vec(state) * 2.0f - glm::vec3(1.0);
            light_position = (objects[lights[light_index]]).transform * glm::vec4(light_position, 1.0);

            auto id = intersectMany(objects, num_objects, hit.position + hit.normal * 0.001f, glm::normalize(light_position - hit.position));

            glm::vec3 light_direction = glm::normalize(id.position - hit.position);

            if (id.hit && id.object == objects + lights[light_index])
            {
                light += id.object->color / (id.t * id.t) * dot(hit.normal, light_direction) * pdf(light_direction, hit.normal, direction, hit.object->roughness);
            }
        }

        accumulated_light += bounced_light * light;

        start = hit.position;
        direction = generateSample(hit.object, direction, hit.normal, state);
    }

    return accumulated_light;
}

__device__ void render(unsigned char *image, glm::mat4 view_matrix, Object *objects, size_t num_objects, int x, int y, int width, int height, hiprandState *state)
{

    double aspect_ratio = (float)width / height;

    double x_scaled = ((float)x / (float)width - 0.5) * 2 * aspect_ratio;
    double y_scaled = ((float)y / (float)height - 0.5) * 2;

    glm::vec3 direction = glm::normalize(glm::mat3(glm::inverse(view_matrix)) * glm::vec3(-x_scaled, -y_scaled, 1.0));
    glm::vec3 start = glm::vec3(glm::inverse(view_matrix)[3]);

    glm::vec3 color = glm::vec3(0.0);

    size_t lights[8];
    size_t num_lights = 0;

    for (int i = 0; i < num_objects; i++)
    {

        if (objects[i].is_light)
        {

            lights[num_lights] = i;
            num_lights++;
        }
    }

    for (int i = 0; i < 4; i++)
    {
        color += trace(start, direction, objects, lights, num_lights, num_objects, state);
    }

    setColor(image, glm::vec4(color / 4.0f, 1.0), x, y, width, height);
}

__global__ void makeWhite(unsigned char *image, glm::mat4 view_matrix, Object *objects, size_t num_objects, int width, int height, int numBlocks, hiprandState *states)
{

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    int id = x + y * (numBlocks * blockDim.x);

    if (x > width || y > height)
        return;

    render(image, view_matrix, objects, num_objects, x, y, width, height, states + id);
}

int main(int argc, char **argv)
{

    hipSetDevice(0);

    sf::Clock clock;
    int width = 1280;
    int height = 720;

    sf::RenderWindow window(sf::VideoMode(width, height), "Hello!");
    sf::Texture render_texture;
    render_texture.create(width, height);

    window.clear(sf::Color(255, 255, 255, 255));
    window.display();

    sf::Sprite image(render_texture);
    image.setPosition(sf::Vector2f(width / 2, height / 2));
    image.setOrigin(sf::Vector2f(render_texture.getSize()) / 2.f);

    render_texture.update(window);

    size_t image_size = width * height * sizeof(char) * 4;

    // Do something
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x, (height + threadsperBlock.y - 1) / threadsperBlock.y);

    unsigned char *h_image = new unsigned char[image_size];
    unsigned char *d_image;
    hipMalloc(&d_image, image_size);

    glm::mat4 view_matrix = glm::lookAtLH(glm::vec3(0.0, 0.0, -3.0), glm::vec3(0.0), glm::vec3(0.0, 1.0, 0.0));
    // Goes from World to local

    float _;
    glm::vec3 _2;

    static const size_t num_objects = 4;

    Object *objects = new Object[num_objects];

    objects[0] = Object{
        glm::mat4(1.0),
        glm::vec3(1.0, 0.0, 0.0),
        0.1,
        false,
        ObjectType::Sphere};
    objects[1] = Object{
        glm::scale(glm::translate(glm::mat4(1.0), glm::vec3(0.0, -1.0, 0.0)), glm::vec3(10.0f)),
        glm::vec3(1.0),
        1.0,
        false,
        ObjectType::Plane};
    objects[2] = Object{
        glm::translate(glm::mat4(1.0), glm::vec3(0.0, 4.0, 0.0)),
        glm::vec3(1.0, 1.0, 1.0) * 10.0f,
        1.0,
        true,
        ObjectType::Sphere};
    objects[3] = Object{
        glm::inverse(glm::lookAtLH(glm::vec3(-2.0, 2.0, -2.0), glm::vec3(0.0), glm::vec3(0.0, 1.0, 0.0))),
        glm::vec3(1.0),
        0.001,
        false,
        ObjectType::Plane};

    Object *d_objects;
    hipMalloc(&d_objects, sizeof(Object) * num_objects);

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState) * numBlocks.x * threadsperBlock.x * numBlocks.y * threadsperBlock.y);

    printf("Init random state");
    init_curand_state<<<numBlocks, threadsperBlock>>>(d_state, numBlocks.x, 1234);
    printf("Done");

    hipMemcpy(d_objects, objects, num_objects * sizeof(Object), hipMemcpyHostToDevice);

    sf::Vector2i mouse_delta;

    sf::Clock fps_timer;
    double last_time = 0.0;

    sf::Font font;
    font.loadFromFile("/usr/share/fonts/opentype/roboto/slab/RobotoSlab-Blod.otd");

    window.setMouseCursorGrabbed(true);
    sf::Mouse::setPosition(window.getPosition() + sf::Vector2i(window.getSize()) / 2);

    // Print view matrix
    printf("%f %f %f %f\n", view_matrix[0][0], view_matrix[0][1], view_matrix[0][2], view_matrix[0][3]);
    printf("%f %f %f %f\n", view_matrix[1][0], view_matrix[1][1], view_matrix[1][2], view_matrix[1][3]);
    printf("%f %f %f %f\n", view_matrix[2][0], view_matrix[2][1], view_matrix[2][2], view_matrix[2][3]);
    printf("%f %f %f %f\n", view_matrix[3][0], view_matrix[3][1], view_matrix[3][2], view_matrix[3][3]);

    while (window.isOpen())
    {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();

            if (event.type == sf::Event::MouseMoved)
            {
            }
        }

        if (window.hasFocus())
        {
            sf::Vector2i center(window.getSize().x / 2, window.getSize().y / 2);
            mouse_delta = sf::Mouse::getPosition(window) - center;
            sf::Mouse::setPosition(center, window);

            glm::mat4 rotation = glm::mat4(glm::rotate(glm::mat4(1.0), mouse_delta.y / 100.f, glm::vec3(1.0, 0.0, 0.0)));
            rotation = glm::mat4(glm::rotate(rotation, mouse_delta.x / 100.f, glm::vec3(0.0, 1.0, 0.0)));

            view_matrix = rotation * view_matrix;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::W))
        {
            view_matrix = glm::translate(view_matrix, glm::inverse(glm::mat3(view_matrix)) * glm::vec3(0.0, 0.0, -0.01));
        }

        window.clear();

        // std::cout << 1.0f / (float)last_time << std::endl;

        makeWhite<<<numBlocks, threadsperBlock>>>(d_image, view_matrix, d_objects, num_objects, width, height, numBlocks.x, d_state);

        // for (int x = 0; x < width; ++x)
        // {
        //     for (int y = 0; y < height; ++y)
        //     {
        //         render(h_image, view_matrix, objects, num_objects, x, y, width, height, 0.0);
        //     }
        // }

        hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);

        // memset(h_image, (unsigned char)255, image_size);

        render_texture.update(h_image);

        window.draw(image);

        window.display();

        last_time = fps_timer.getElapsedTime().asSeconds();
        fps_timer.restart();
    }

    return 0;
}
